
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloCUDA()
{
    int threadID = threadIdx.x; // Modifica la forma en que se calcula el identificador del hilo
    printf("Hello, World! I'm thread %d. Nuestro nombre es Mario de León y Javier Valle y nuestros carnets son 19019 y 20159\n", threadID);
}

int main()
{
    int numBlocks = 1;          // Número de bloques (1 bloque)
    int threadsPerBlock = 1024; // Número de hilos por bloque (2048 hilos)

    // Llama al kernel con la configuración de bloques y hilos
    helloCUDA<<<numBlocks, threadsPerBlock>>>();
    helloCUDA<<<numBlocks, threadsPerBlock>>>();

    // Espera a que todos los hilos hayan terminado
    hipDeviceSynchronize();

    return 0;
}
